#include <cuda/Cuda.hpp>
#include "hip/hip_runtime.h"  
#include ""  
#include "hip/hip_runtime_api.h"
#include <helper_functions.h>

namespace ORB_SLAM2 { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
